#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

// for 1024 threads per block
#define BLOCK_WIDTH 32   
#define BLOCK_HEIGHT 32   

__global__ void find_min_max(
        const float* const d_logLuminance,
        float* d_limits,
        const unsigned int count)
{
    float __min = d_logLuminance[count-1];
    float __max = __min;
    const unsigned int tid = threadIdx.x;
    extern __shared__ float sdata[];

    for (unsigned int base = 0; base < count; base += blockDim.x)
    {
        if (tid+base >= count)
            return;
        sdata[tid] = d_logLuminance[base + tid];
        __syncthreads();
        for (unsigned int s = blockDim.x / 2; s; s /= 2)
        {
            if (tid<s)
            {
                float _max = max(
                    sdata[blockDim.x - 1 - tid],
                    sdata[blockDim.x - 1 - tid - s]
                );
                sdata[tid] = min(sdata[tid], sdata[tid+s]);
                sdata[blockDim.x - 1 - tid] = _max;
            }
            // take uneven case into account
            if (s&1 && s!=1)
                s++;
            __syncthreads();
        }
        __syncthreads();
        if (tid==0)
        {
            if (__min > sdata[0])
                __min = sdata[0];
            if (__max < sdata[blockDim.x - 1])
                __max = sdata[blockDim.x - 1];
        }
    }
    if (tid==0)
    {
        d_limits[0] = __min;
        d_limits[1] = __max;
    }
}

__global__ void calc_hist_grid(const float* const d_logLuminance, 
                               const float* d_min,
                               const float* d_max,
                               unsigned int *histogram,
                               const size_t numRows,
                               const size_t numCols,
                               const size_t numBins)
{
    extern __shared__ float s_histogram[];
    float lumRange = d_max[0]-d_min[0];
    
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

    // 0) init variables
    const dim3 blockSize = dim3(BLOCK_WIDTH, BLOCK_HEIGHT);
    const dim3 gridSize = dim3(1+numCols/BLOCK_WIDTH, 1+numRows/BLOCK_HEIGHT);

    float *d_limits;
    unsigned int *histogram;
    checkCudaErrors(hipMalloc(&d_limits, sizeof(float) * 2));
    checkCudaErrors(hipMalloc(&histogram, sizeof(unsigned int) * numBins));
    float* h_limits = (float*)malloc(sizeof(float) * 2);

    // 1) find the minimum and maximum value in the input logLuminance channel
    find_min_max<<<1, BLOCK_WIDTH * BLOCK_HEIGHT, sizeof(float) * BLOCK_WIDTH * BLOCK_HEIGHT>>>
        (d_logLuminance, d_limits, numRows * numCols);

    checkCudaErrors(hipMemcpy(h_limits, d_limits, sizeof(float) * 2, hipMemcpyDeviceToHost));

    printf("_min = %f, _max = %f\n", h_limits[0], h_limits[1]);

    // 2) in-place
    // 3) Calc histogram with per thread atomic (1st step)
//    calc_hist_grid<<<gridSize, blockSize, sizeof(unsigned int) * gridSize.x * gridSize.y>>>
//        (d_logLuminance, d_min, d_max, histogram, numRows, numCols, numBins);
    //    reduce grid of histograms into single one

}
